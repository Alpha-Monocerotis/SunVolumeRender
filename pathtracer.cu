#include "hip/hip_runtime.h"
//
// Created by 孙万捷 on 16/3/4.
//

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

#include "utils/hip/hip_runtime_api.h"
#include "core/cuda_bbox.h"
#include "core/cuda_camera.h"
#include "core/cuda_transfer_function.h"
#include "common.h"

// global variables
__constant__ cudaTransferFunction transferFunction;

extern "C" void setup_transferfunction(const cudaTransferFunction& tf)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(transferFunction), &tf, sizeof(cudaTransferFunction), 0));
}

__host__ __device__ unsigned int wangHash(unsigned int a)
{
    a = (a ^ 61) ^ (a >> 16);
    a = a + (a << 3);
    a = a ^ (a >> 4);
    a = a * 0x27d4eb2d;
    a = a ^ (a >> 15);

    return a;
}

template <typename T>
__global__ void clear_hdr_buffer(T* buffer)
{
    auto idx = blockDim.x * blockIdx.x + threadIdx.x;
    auto idy = blockDim.y * blockIdx.y + threadIdx.y;
    auto offset = idy * WIDTH + idx;

    buffer[offset] = T(0.f);
}

__global__ void render_kernel(glm::u8vec4* img, const cudaBBox volumeBox, const cudaCamera camera, unsigned int hashedFrameNo)
{
    auto idx = blockDim.x * blockIdx.x + threadIdx.x;
    auto idy = blockDim.y * blockIdx.y + threadIdx.y;
    auto offset = idy * WIDTH + idx;
    hiprandState rng;
    hiprand_init(hashedFrameNo + offset, 0, 0, &rng);

    cudaRay ray;
    camera.GenerateRay(idx, idy, rng, &ray);

    float tNear, tFar;
    if(!volumeBox.Intersect(ray, &tNear, &tFar))
        img[offset] = glm::u8vec4(0, 0, 0, 0);
    else
        img[offset] = glm::u8vec4(255, 0, 0, 255);
}

extern "C" void rendering(glm::u8vec4* img, const cudaBBox& volumeBox, const cudaCamera& camera, unsigned int frameNo)
{
    dim3 blockSize(16, 16);
    dim3 gridSize(WIDTH / blockSize.x, HEIGHT / blockSize.y);

    render_kernel<<<gridSize, blockSize>>>(img, volumeBox, camera, frameNo);
}