#include "hip/hip_runtime.h"
//
// Created by 孙万捷 on 16/3/4.
//

#include <stdio.h>

#define GLM_FORCE_NO_CTOR_INIT
#define GLM_FORCE_INLINE
#include <glm/glm.hpp>

#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

#include "utils/hip/hip_runtime_api.h"
#include "core/cuda_camera.h"
#include "core/cuda_transfer_function.h"
#include "core/cuda_volume.h"
#include "core/render_parameters.h"
#include "core/tonemapping.h"
#include "core/woodcock_tracking.h"
#include "core/transmittance.h"
#include "core/bsdf/henyey_greenstein.h"
#include "core/bsdf/lambert.h"
#include "core/bsdf/microfacet.h"
#include "core/lights/lights.h"
#include "core/lights/light_sample.h"

#define PHASE_FUNC_G (0.f)
#define IOR (2.5f)
#define ALPHA (0.15f)

// global variables
__constant__ cudaTransferFunction transferFunction;
extern "C" void setup_transferfunction(const cudaTransferFunction& tf)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(transferFunction), &tf, sizeof(cudaTransferFunction), 0));
    checkCudaErrors(hipDeviceSynchronize());
}

__constant__ cudaVolume volume;
extern "C" void setup_volume(const cudaVolume& vol)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(volume), &vol, sizeof(cudaVolume), 0));
    checkCudaErrors(hipDeviceSynchronize());
}

__constant__ cudaCamera camera;
extern "C" void setup_camera(const cudaCamera& cam)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(camera), &cam, sizeof(cudaCamera), 0));
    checkCudaErrors(hipDeviceSynchronize());
}

__constant__ uint32_t num_areaLights;
__constant__ cudaAreaLight areaLights[MAX_LIGHT_SOURCES];
extern "C" void setup_area_lights(cudaAreaLight* lights, uint32_t n)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(num_areaLights), &n, sizeof(uint32_t), 0));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(areaLights), lights, sizeof(cudaAreaLight) * n, 0));
}

__constant__ cudaEnvironmentLight envLight;
extern "C" void setup_env_lights(const cudaEnvironmentLight& light)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(envLight), &light, sizeof(cudaEnvironmentLight), 0));
    checkCudaErrors(hipDeviceSynchronize());
}

__host__ __device__ uint32_t wangHash(uint32_t a)
{
    a = (a ^ 61) ^ (a >> 16);
    a = a + (a << 3);
    a = a ^ (a >> 4);
    a = a * 0x27d4eb2d;
    a = a ^ (a >> 15);

    return a;
}

__inline__ __device__ void running_estimate(glm::vec3& acc_buffer, const glm::vec3& curr_est, unsigned int N)
{
    acc_buffer += (curr_est - acc_buffer) / (N + 1.f);
}

template <typename T>
__global__ void clear_hdr_buffer(T* buffer)
{
    auto idx = blockDim.x * blockIdx.x + threadIdx.x;
    auto idy = blockDim.y * blockIdx.y + threadIdx.y;
    auto offset = idy * WIDTH + idx;

    buffer[offset] = T(0.f);
}

__inline__ __device__ bool terminate_with_raussian_roulette(glm::vec3* troughput, hiprandState& rng)
{
    float illum = 0.2126f * troughput->x + 0.7152f * troughput->y + 0.0722 * troughput->z;
    if(hiprand_uniform(&rng) > illum) return true;
    *troughput /= illum;

    return false;
}

enum ShadingType{SHANDING_TYPE_ISOTROPIC, SHANDING_TYPE_BRDF};
__inline__ __device__ glm::vec3 bsdf(const VolumeSample& vs, const glm::vec3& wi, ShadingType st)
{
    glm::vec3 diffuseColor = glm::vec3(vs.color_opacity.x, vs.color_opacity.y, vs.color_opacity.z);

    glm::vec3 L;
    if(st == SHANDING_TYPE_ISOTROPIC)
    {
         L = diffuseColor * hg_phase_f(vs.wo, wi);
    }
    else if(st == SHANDING_TYPE_BRDF)
    {
        auto normal = glm::normalize(vs.gradient);
        normal = glm::dot(vs.wo, normal) < 0.f ? -normal : normal;

        float cosTerm = fmaxf(0.f, glm::dot(wi, normal));
        float ks = schlick_fresnel(1.0f, IOR, cosTerm);
        float kd = 1.f - ks;

        auto diffuse = diffuseColor * lambert_brdf_f(wi, vs.wo);
        auto specular = glm::vec3(1.f) * microfacet_brdf_f(wi, vs.wo, normal, IOR, ALPHA);

        L = (kd * diffuse + ks * specular) * cosTerm;
    }

    return L;
}

__inline__ __device__ glm::vec3 sample_bsdf(const VolumeSample& vs, glm::vec3* wi, float* pdf, hiprandState& rng, ShadingType st)
{
    if(st == SHANDING_TYPE_ISOTROPIC)
    {
        hg_phase_sample_f(PHASE_FUNC_G, vs.wo, wi, pdf, rng);
        return glm::vec3(vs.color_opacity) * hg_phase_f(vs.wo, *wi);
    }
    else if(st == SHANDING_TYPE_BRDF)
    {
        auto normal = glm::normalize(vs.gradient);
        auto cosTerm = glm::dot(vs.wo, normal);
        if(cosTerm < 0.f)
        {
            cosTerm = -cosTerm;
            normal = -normal;
        }

        auto ks = schlick_fresnel(1.f, IOR, cosTerm);
        auto kd = 1.f - ks;
        auto p = 0.25f + 0.5f * ks;

        if(hiprand_uniform(&rng) < p)
        {
            microfacet_brdf_sample_f(vs.wo, normal, ALPHA, wi, pdf, rng);
            auto f = microfacet_brdf_f(*wi, vs.wo, normal, IOR, ALPHA);
            return glm::vec3(1.f) * f * ks / p;
        }
        else
        {
            lambert_brdf_sample_f(vs.wo, normal, wi, pdf, rng);
            auto f = lambert_brdf_f(*wi, vs.wo);
            return glm::vec3(vs.color_opacity.x, vs.color_opacity.y, vs.color_opacity.z) * f * kd / (1.f - p);
        }
    }

    return glm::vec3(0.f);
}

__inline__ __device__ glm::vec3 estimate_direct_light(const VolumeSample vs, hiprandState& rng, ShadingType st)
{
    glm::vec3 Li = glm::vec3(0.f);

    if(num_areaLights == 0)
        return Li;

    // randomly choose a single light
    int lightId = num_areaLights * hiprand_uniform(&rng);
    lightId = lightId < num_areaLights ? lightId : num_areaLights - 1;
    const cudaAreaLight& light = areaLights[lightId];

    // sample light
    glm::vec3 lightPos;
    glm::vec3 wi;
    float pdf;
    Li = sample_light(light, vs.ptInWorld, rng, &lightPos, &wi, &pdf);

    if(pdf > 0.f && fmaxf(Li.x, fmaxf(Li.y, Li.z)) > 0.f)
    {
        auto Tr = transmittance(vs.ptInWorld, lightPos, volume, transferFunction, rng);
        Li = Tr * num_areaLights * bsdf(vs, wi, st) * Li / pdf;
    }
    else
        Li = glm::vec3(0.f);

    return Li;
}

__global__ void kernel_pathtracer(const RenderParams renderParams, uint32_t hashedFrameNo)
{
    auto idx = blockDim.x * blockIdx.x + threadIdx.x;
    auto idy = blockDim.y * blockIdx.y + threadIdx.y;
    auto offset = idy * WIDTH + idx;
    hiprandState rng;
    hiprand_init(hashedFrameNo + offset, 0, 0, &rng);

    glm::vec3 L = glm::vec3(0.f);
    glm::vec3 T = glm::vec3(1.f);

    cudaRay ray;
    camera.GenerateRay(idx, idy, rng, &ray);

    LightSample ls;
    bool hitLight = get_nearest_light_sample(ray, areaLights, num_areaLights, &ls);
    for(auto k = 0; k < renderParams.traceDepth; ++k)
    {
        auto t = sample_distance(ray, volume, transferFunction, rng);

        if((k == 0) && hitLight)
        {
            t = t < 0.f ? FLT_MAX : t;
            if(ls.t < t)
            {
                auto cosTerm = glm::dot(ls.normal, -ray.dir);
                L += T * ls.radiance * (cosTerm <= 0.f ? 0.f : 1.f);
                break;
            }
        }

        if(t < 0.f)
        {
            //L += T * envLight.GetEnvRadiance(ray.dir);
            break;
        }

        VolumeSample vs;

        vs.wo = -ray.dir;
        vs.ptInWorld = ray.PointOnRay(t);
        vs.intensity = volume(vs.ptInWorld);
        vs.color_opacity = transferFunction(vs.intensity);
        vs.gradient = volume.Gradient_CentralDiff(vs.ptInWorld);
        vs.gradientMagnitude = sqrtf(glm::dot(vs.gradient, vs.gradient));

        glm::vec3 wi;
        float pdf = 0.f;
        ShadingType st;
        if(vs.gradientMagnitude < 1e-3)
            st = SHANDING_TYPE_ISOTROPIC;
        else
            st = SHANDING_TYPE_BRDF;

        L += T * estimate_direct_light(vs, rng, st);

        auto f = sample_bsdf(vs, &wi, &pdf, rng, st);
        float cosTerm = fabsf(glm::dot(glm::normalize(vs.gradient), wi));
        if(fmaxf(f.x, fmaxf(f.y, f.z)) > 0.f && pdf > 0.f)
        {
            if(st == SHANDING_TYPE_ISOTROPIC)
                T *= f / pdf;
            else
                T *= f * cosTerm / pdf;
        }

        ray.orig = vs.ptInWorld;
        ray.dir = wi;

        if(k >= 3)
        {
            if(terminate_with_raussian_roulette(&T, rng))
                break;
        }
    }

    running_estimate(renderParams.hdrBuffer[offset], L, renderParams.frameNo);
}

__global__ void hdr_to_ldr(glm::u8vec4* img, const RenderParams renderParams)
{
    auto idx = blockDim.x * blockIdx.x + threadIdx.x;
    auto idy = blockDim.y * blockIdx.y + threadIdx.y;
    auto offset = idy * WIDTH + idx;

    auto L = reinhard_tone_mapping(renderParams.hdrBuffer[offset], camera.exposure);
    img[offset] = glm::u8vec4(L.x * 255, L.y * 255, L.z * 255, 255);
}

extern "C" void render_pathtracer(glm::u8vec4* img, const RenderParams& renderParams)
{
    dim3 blockSize(16, 16);
    dim3 gridSize(WIDTH / blockSize.x, HEIGHT / blockSize.y);

    if(renderParams.frameNo == 0)
    {
        clear_hdr_buffer<<<gridSize, blockSize>>>(renderParams.hdrBuffer);
    }

    kernel_pathtracer<<<gridSize, blockSize>>>(renderParams, wangHash(renderParams.frameNo));
    hdr_to_ldr<<<gridSize, blockSize>>>(img, renderParams);
}