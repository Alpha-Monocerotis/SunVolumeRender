#include "hip/hip_runtime.h"
#define GLM_FORCE_NO_CTOR_INIT
#define GLM_FORCE_INLINE
#include <glm/glm.hpp>

#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

#include "common.h"
#include "utils/hip/hip_runtime_api.h"
#include "core/cuda_camera.h"
#include "core/cuda_transfer_function.h"
#include "core/cuda_volume.h"

__global__ void kernel_raycasting(glm::u8vec4* img, cudaVolume volume, cudaTransferFunction transferFunction, cudaCamera camera, float stepSize)
{
    auto idx = blockDim.x * blockIdx.x + threadIdx.x;
    auto idy = blockDim.y * blockIdx.y + threadIdx.y;
    auto offset = idy * WIDTH + idx;

    cudaRay ray;
    camera.GenerateRay(idx, idy, &ray);

    glm::vec4 L = glm::vec4(0.f);

    float tNear, tFar, t;
    if(volume.Intersect(ray, &tNear, &tFar))
    {
        t = tNear;
        while(t <= tFar)
        {
            auto ptInWorld = ray.PointOnRay(t);
            auto intensity = volume(ptInWorld);
            auto color_opacity = transferFunction(intensity);

            // apply lighting
            auto gradient = volume.Gradient_CentralDiff(ptInWorld);
            auto gradientMagnitude = sqrtf(glm::dot(gradient, gradient));
            float cosTerm = 1.f;
            float specularTerm = 0.f;
            if(gradientMagnitude > 1e-3)
            {
                auto normal = glm::normalize(gradient);
                auto lightDir = glm::normalize(camera.pos - ptInWorld);
                cosTerm = fabsf(glm::dot(normal, lightDir));

                specularTerm = powf(cosTerm, 30.f);
            }

            color_opacity.x = color_opacity.x * color_opacity.w * cosTerm * 0.8f + color_opacity.w * specularTerm * 0.2f;
            color_opacity.y = color_opacity.y * color_opacity.w * cosTerm * 0.8f + color_opacity.w * specularTerm * 0.2f;
            color_opacity.z = color_opacity.z * color_opacity.w * cosTerm * 0.8f + color_opacity.w * specularTerm * 0.2f;

            L += (1.f - L.w) * color_opacity;

            if(L.w > 0.95f) break;

            t += stepSize * 0.5f;
        }

    }

    L.x = fminf(L.x, 1.f);
    L.y = fminf(L.y, 1.f);
    L.z = fminf(L.z, 1.f);
    img[offset] = glm::u8vec4(L.x * 255, L.y * 255, L.z * 255, 255 * L.w);
}

extern "C" void render_raycasting(glm::u8vec4* img, cudaVolume& volume, cudaTransferFunction& transferFunction, cudaCamera& camera, float stepSize)
{
    dim3 blockSize(16, 16);
    dim3 gridSize(WIDTH / blockSize.x, HEIGHT / blockSize.y);

    kernel_raycasting<<<gridSize, blockSize>>>(img, volume, transferFunction, camera, stepSize);
}